#include <iostream>
#include <cmath>
#include <hip/hip_runtime.h>
#include <hipfft/hipfft.h>

const int N = 1024; // 图像大小
const int BATCH = 1; // 批处理大小

// CUDA FFT 执行函数
void fft_cuda(hipfftComplex* data, int width, int height) {
    hipfftHandle plan;
    // 创建 FFT 计划
    hipfftPlan2d(&plan, width, height, HIPFFT_C2C);

    // 执行 FFT
    hipfftExecC2C(plan, data, data, HIPFFT_FORWARD);

    // 销毁 FFT 计划
    hipfftDestroy(plan);
}

int main() {
    // 分配内存并初始化数据
    hipfftComplex* data;
    hipMallocManaged(&data, N * N * BATCH * sizeof(hipfftComplex));
    for (int i = 0; i < N * N * BATCH; ++i) {
        data[i].x = sin(2 * M_PI * i / (N * N));
        data[i].y = 0.0f;
    }

    // 测量优化前的运行时间
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start);

    // 执行 FFT
    fft_cuda(data, N, N);

    hipEventRecord(stop);
    hipEventSynchronize(stop);
    float milliseconds = 0;
    hipEventElapsedTime(&milliseconds, start, stop);
    std::cout << "运行时间（优化）: " << milliseconds << " 毫秒" << std::endl;

    // 释放内存
    hipFree(data);

    return 0;
}
